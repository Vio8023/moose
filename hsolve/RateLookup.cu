#include "hip/hip_runtime.h"
/**********************************************************************
** This program is part of 'MOOSE', the
** Messaging Object Oriented Simulation Environment.
**   copyright (C) 2003-2007 Upinder S. Bhalla, Niraj Dudani and NCBS
** It is made available under the terms of the
** GNU Lesser General Public License version 2.1
** See the file COPYING.LIB for the full notice.
**********************************************************************/
#define USE_CUDA
#include <vector>
#include <stdio.h>
using namespace std;

#include "RateLookup.h"

#ifdef USE_CUDA
#define BLOCK_WIDTH 256
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#endif


LookupTable::LookupTable(
	double min, double max, unsigned int nDivs, unsigned int nSpecies )
{
	min_ = min;
	max_ = max;
	// Number of points is 1 more than number of divisions.
	// Then add one more since we may interpolate at the last point in the table.
	nPts_ = nDivs + 1 + 1;
	dx_ = ( max - min ) / nDivs;
	// Every row has 2 entries for each type of gate
	nColumns_ = 2 * nSpecies;
	
	//~ interpolate_.resize( nSpecies );
	table_.resize( nPts_ * nColumns_ );
	
}

void LookupTable::addColumns(
	int species,
	const vector< double >& C1,
	const vector< double >& C2 )
	//~ const vector< double >& C2,
	//~ bool interpolate )
{
	vector< double >::const_iterator ic1 = C1.begin();
	vector< double >::const_iterator ic2 = C2.begin();
	vector< double >::iterator iTable = table_.begin() + 2 * species;
	// Loop until last but one point
	for ( unsigned int igrid = 0; igrid < nPts_ - 1 ; ++igrid ) {
		*( iTable )     = *ic1;
		*( iTable + 1 ) = *ic2;
		
		iTable += nColumns_;
		++ic1, ++ic2;
	}
	// Then duplicate the last point
	*( iTable )     = C1.back();
	*( iTable + 1 ) = C2.back();
	
	//~ interpolate_[ species ] = interpolate;
}

void LookupTable::column( unsigned int species, LookupColumn& column )
{
	column.column = 2 * species;
	//~ column.interpolate = interpolate_[ species ];
}

void LookupTable::row( double x, LookupRow& row )
{
	if ( x < min_ )
		x = min_;
	else if ( x > max_ )
		x = max_;
	
	double div = ( x - min_ ) / dx_;
	unsigned int integer = ( unsigned int )( div );
	
	row.fraction = div - integer;
	row.row = &( table_.front() ) + integer * nColumns_;
}

#ifdef USE_CUDA

__global__
void
row_kernel(double * d_x, 
		   LookupRow * d_row, 
		   double min,
		   double max, 
		   double dx,
		   unsigned int nColumns, 
		   unsigned int size,
		   double * address)
{
			   
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
	if(tid >= size) return;
	
	//if(tid == 0) printf("kernel launch successful!\n");
	
	double x = d_x[tid];
	LookupRow row = d_row[tid];
	
	if ( x < min )
		x = min;
	else if ( x > max )
		x = max;
	
	double div = ( x - min ) / dx;
	unsigned int integer = ( unsigned int )( div );
	
	row.fraction = div - integer;
	row.row = address +  integer * nColumns;	
	
}

void LookupTable::row_gpu(vector<double>::iterator& x, vector<LookupRow>::iterator& row, unsigned int size){
	
	//printf("start row_gpu calculation...\n");
	
	thrust::device_vector<double> d_x(size);
	thrust::device_vector<LookupRow> d_row(size);
	thrust::copy(x, x+size, d_x.begin());
	thrust::copy(row, row+size, d_row.begin());
	
	double * d_x_p = thrust::raw_pointer_cast(d_x.data());
	LookupRow * d_row_p = thrust::raw_pointer_cast(d_row.data());
	
    const dim3 gridSize(size/BLOCK_WIDTH + 1, 1, 1);
    const dim3 blockSize(BLOCK_WIDTH,1,1);
    
    row_kernel<<<gridSize, blockSize>>>(d_x_p, d_row_p, min_, max_, dx_, nColumns_, size, &table_.front());	
    
    hipDeviceSynchronize();
    hipDeviceSynchronize(); 
    
    //printf("kernel launch finished...\n");
    LookupRow * h_row;
    h_row = (LookupRow *) malloc(sizeof(LookupRow)*size);
    hipMemcpy(h_row, d_row_p, sizeof(LookupRow)*size, hipMemcpyDeviceToHost);
    std::copy(h_row, h_row+size, row);
    
    //printf("finish row_gpu calculation...\n");
}
#endif

void LookupTable::lookup(
	const LookupColumn& column,
	const LookupRow& row,
	double& C1,
	double& C2 )
{
	double a, b;
	double *ap, *bp;
	
	ap = row.row + column.column;
	
	//~ if ( ! column.interpolate ) {
		//~ C1 = *ap;
		//~ C2 = *( ap + 1 );
		//~ 
		//~ return;
	//~ }
	
	bp = ap + nColumns_;
	
	a = *ap;
	b = *bp;
	C1 = a + ( b - a ) * row.fraction;
	
	a = *( ap + 1 );
	b = *( bp + 1 );
	C2 = a + ( b - a ) * row.fraction;
}
